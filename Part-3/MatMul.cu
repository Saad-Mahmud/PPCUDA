
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;

// Kernel function to add the elements of two arrays
__global__
void GPUmatmul(int N, double *x, double *y, double *ans)
{
    int t= (blockDim.x*blockDim.y)*threadIdx.z+(threadIdx.y*blockDim.x)+(threadIdx.x);
    int b= (gridDim.x*gridDim.y)*blockIdx.z+(blockIdx.y*gridDim.x)+(blockIdx.x);
    int T= blockDim.x*blockDim.y*blockDim.z;
    int B= gridDim.x*gridDim.y*gridDim.z;
    
    for (int i=b;i<N;i+=B)
    {
		for(int j=t;j<N;j+=T)
		{
			for(int k=0;k<N;k++)
			{
				ans[i*N+j]+=(x[i*N+k]*y[k*N+j]);
			}
		}
	}
}

void CPUmatmul(int N,double *x, double *y, double *ans)
{
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<N;j++)
		{
			for(int k=0;k<N;k++)
			{
				ans[i*N+j]+=(x[i*N+k]*y[k*N+j]);
			}
		}
	}
}

bool sanity_check(int N,double *ans)
{
	for(int i=0;i<N;i++)
	{
		for(int j=0;j<N;j++)
		{
			if(ans[i*N+j]!=20.0)return false;
		}
	}
	return true;
}

void printtt(int N,double *ans)
{
	for(int i=0;i<16;i++)
	{
		for(int j=0;j<16;j++)
		{
			cout<<ans[i*N+j]<<" ";
		}
		cout<<endl;
	}
}
int main(void)
{

	int N = 1<<13;
	double *x, *y, *ans;

	// Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged(&x, N*N*sizeof(double));
	hipMallocManaged(&y, N*N*sizeof(double));
	hipMallocManaged(&ans, N*N*sizeof(double));

	// initialize x,y and ans arrays on the host
	for (int i = 0; i < N; i++) 
	{
		for(int j=0;j<N;j++)
		{
			x[i*N+j]=5;
			y[i*N+j]=(i==j?1:0);
			ans[i*N+j]=(double)0.000000000000;
		}
	}

  
  clock_t t;
  double avg=0;
  // Run kernel on 1M elements on the GPU
  for(int i=0;i<=3;i++)
  {
	  t=clock();
	  //CPUmatmul(N, x, y,ans);
	  t = clock() - t;
	  if(i)avg+=t;
	  printf ("It took CPU-%d %f ms.\n",i,(((double)t)/CLOCKS_PER_SEC)*1000);
  }
  avg/=3;
  avg/=CLOCKS_PER_SEC;
  avg*=1000;
  printf ("It took %lf ms on avg.\n",avg);
  if(sanity_check(N,ans))cout<<"RUN OK."<<endl;
  else cout<<"RUN NOT OK."<<endl;
 
	// initialize x,y and ans arrays on the host
	for (int i = 0; i < N; i++) 
	{
		for(int j=0;j<N;j++)
		{
			x[i*N+j]=5;
			y[i*N+j]=(i==j?1:0);
			ans[i*N+j]=(double)0.000000000000;
		}
	}
   avg=0;
  // Run kernel on 1M elements on the GPU
  for(int i=0;i<=3;i++)
  {
	  t=clock();
	  GPUmatmul<<<dim3(16,16,16), dim3(16,8,8)>>>(N, x, y,ans);
	  hipDeviceSynchronize();
	  t = clock() - t;
	  if(i)avg+=t;
	  printf ("It took GPU-%d %f ms.\n",i,(((double)t)/CLOCKS_PER_SEC)*1000);
  }
  avg/=3;
  avg/=CLOCKS_PER_SEC;
  avg*=1000;
  printf ("It took %lf ms on avg.\n",avg);
  if(sanity_check(N,ans))cout<<"RUN OK."<<endl;
  else cout<<"RUN NOT OK."<<endl;

  
  // Free memory
  hipFree(x);
  hipFree(y); 
  return 0;
}



